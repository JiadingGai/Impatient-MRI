#include "hip/hip_runtime.h"
/**************************************************************************
   (C) Copyright 2010 The Board of Trustees of the University of Illinois.
   All rights reserved.

   Developed by:

                    IMPACT & MRFIL Research Groups
               University of Illinois, Urbana Champaign

   Permission is hereby granted, free of charge, to any person obtaining a copy
   of this software and associated documentation files (the "Software"), to
   deal with the Software without restriction, including without limitation the
   rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
   sell copies of the Software, and to permit persons to whom the Software is
   furnished to do so, subject to the following conditions:

   Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimers.

   Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimers in the documentation
   and/or other materials provided with the distribution.

   Neither the names of the IMPACT Research Group, MRFIL Research Group, the
   University of Illinois, nor the names of its contributors may be used to
   endorse or promote products derived from this Software without specific
   prior written permission.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
   OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
   CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
   THE SOFTWARE.
 *****************************************************************************/

/*****************************************************************************

    File Name   [Dverti2dGpu.cu]

    Revision    [0.1; Initial build; Fan Lam, Mao-Jing Fu, ECE UIUC]
    Date        [10/25/2010]

*****************************************************************************/

/*---------------------------------------------------------------------------*/
/*  Included library headers                                                 */
/*---------------------------------------------------------------------------*/

// System libraries
#include <stdio.h>
#include <string.h>

// Project header files
#include <tools.h>
#include <structures.h>

#include <Dverti2dGpu.cuh>

/*---------------------------------------------------------------------------*/
/*  Namespace declared - begin                                               */
/*---------------------------------------------------------------------------*/

//namespace uiuc_mri {

/*---------------------------------------------------------------------------*/
/*  Function definitions                                                     */
/*---------------------------------------------------------------------------*/

/*===========================================================================*/
/*                                                                           */
/*  Synopsis    []                                                           */
/*                                                                           */
/*  Description []                                                           */
/*                                                                           */
/*===========================================================================*/

#define Dverti_BLOCK_X      256
#define Dverti_BLOCK_X_2D   16
#define Dverti_BLOCK_Y_2D   16

__global__ void Dverti2dGpuKernel1(FLOAT_T *s_real, FLOAT_T *s_imag,
           const FLOAT_T *p_real, const FLOAT_T *p_imag,
           const unsigned int num_row, const unsigned int num_col);

__global__ void Dverti2dGpuKernel2(FLOAT_T *s_real, FLOAT_T *s_imag,
           const FLOAT_T *p_real, const FLOAT_T *p_imag,
           const unsigned int num_row, const unsigned int num_col);

    void
Dverti2dGpu(
    FLOAT_T *s_real, FLOAT_T *s_imag,
    const FLOAT_T *p_real, const FLOAT_T *p_imag,
    const unsigned int num_row, const unsigned int num_col)
{
    startMriTimer(getMriTimer()->timer_Dverti2dGpu);

    int num_blocks_x = ceil((FLOAT_T) num_col / (FLOAT_T) Dverti_BLOCK_X);
    int num_blocks_x_2d = ceil((FLOAT_T) num_col / (FLOAT_T) Dverti_BLOCK_X_2D);
    int num_blocks_y_2d = ceil((FLOAT_T) num_row / (FLOAT_T) Dverti_BLOCK_Y_2D);

    makeSure(num_blocks_x <= 65535, "Maximum supported num_col is 64K-1.");
    makeSure(num_blocks_x_2d * num_blocks_y_2d <= 65535,
        "Maximum supported (num_col*num_row) is 64K-1.");

    Dverti2dGpuKernel1 <<<dim3(num_blocks_x_2d, num_blocks_y_2d),
                          dim3(Dverti_BLOCK_X_2D, Dverti_BLOCK_Y_2D)>>>
        (s_real, s_imag, p_real, p_imag, num_row, num_col);

    Dverti2dGpuKernel2 <<<dim3(num_blocks_x), dim3(Dverti_BLOCK_X)>>>
        (s_real, s_imag, p_real, p_imag, num_row, num_col);

    stopMriTimer(getMriTimer()->timer_Dverti2dGpu);
}

__global__ void Dverti2dGpuKernel1(FLOAT_T *s_real, FLOAT_T *s_imag,
           const FLOAT_T *p_real, const FLOAT_T *p_imag,
           const unsigned int num_row, const unsigned int num_col)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    #if 1
    if(j < (num_row-1))
    {
        unsigned int i_num_row_j = i*num_row + j;
        unsigned int i_num_row_j_1 = i_num_row_j + 1;
        s_real[i_num_row_j] = p_real[i_num_row_j] - p_real[i_num_row_j_1];
        s_imag[i_num_row_j] = p_imag[i_num_row_j] - p_imag[i_num_row_j_1];
    }
    #else
    if(j < (num_row-1))
    {
        s_real[i*num_row + j] = p_real[i*num_row + j] -
                                p_real[i*num_row + j + 1];
        s_imag[i*num_row + j] = p_imag[i*num_row + j] - 
                                   p_imag[i*num_row + j + 1];
    }
    #endif
}

__global__ void Dverti2dGpuKernel2(FLOAT_T *s_real, FLOAT_T *s_imag,
           const FLOAT_T *p_real, const FLOAT_T *p_imag,
           const unsigned int num_row, const unsigned int num_col)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    #if 1
    unsigned int i_num_row = i * num_row;
    unsigned int i_num_row_num_row_1 = i_num_row + num_row - 1;
    s_real[i_num_row_num_row_1] = p_real[i_num_row_num_row_1] -
                                  p_real[i_num_row];
    s_imag[i_num_row_num_row_1] = p_imag[i_num_row_num_row_1] -
                                  p_imag[i_num_row];
    #else
    s_real[i*num_row + num_row - 1] = p_real[i*num_row + num_row - 1] -
                                      p_real[i*num_row]
    s_imag[i*num_row + num_row - 1] = p_imag[i*num_row + num_row - 1] -
                                      p_imag[i*num_row];
    #endif
}



/*---------------------------------------------------------------------------*/
/*  Namespace declared - end                                                 */
/*---------------------------------------------------------------------------*/

//}

