/*
(C) Copyright 2010 The Board of Trustees of the University of Illinois.
All rights reserved.

Developed by:

                         IMPACT Research Groups
                University of Illinois, Urbana Champaign

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to
deal with the Software without restriction, including without limitation the
rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
sell copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

Redistributions of source code must retain the above copyright notice, this
list of conditions and the following disclaimers.

Redistributions in binary form must reproduce the above copyright notice,
this list of conditions and the following disclaimers in the documentation
and/or other materials provided with the distribution.

Neither the names of the IMPACT Research Group, the University of Illinois,
nor the names of its contributors may be used to endorse or promote products
derived from this Software without specific prior written permission.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
THE SOFTWARE.
*/

/*****************************************************************************

    File Name   [xcpplib_sdkGpu.cu]

    Synopsis    [Helper functions modified from NVIDIA CUDA SDK.]

    Description [See the corresponding header file for more information.]

    Revision    [0.1; Initial build; Xiao-Long Wu, ECE UIUC]
    Date        [04/08/2010]

 *****************************************************************************/

/*---------------------------------------------------------------------------*/
/*  Included library headers                                                 */
/*---------------------------------------------------------------------------*/

// System libraries
#include <stdlib.h>
#include <stdio.h>

// CUDA libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// XCPPLIB libraries
#include <xcpplib_process.h>

/*---------------------------------------------------------------------------*/
/*  Namespace declared - begin                                               */
/*---------------------------------------------------------------------------*/

namespace xcpplib {

/*---------------------------------------------------------------------------*/
/*  Namespace used                                                           */
/*---------------------------------------------------------------------------*/

/*---------------------------------------------------------------------------*/
/*  Function implementations                                                 */
/*---------------------------------------------------------------------------*/

    void 
__cudaSafeCall(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() Runtime API error at line %i of %s:\n",
            line, file);
        fprintf(stderr, "Error message: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

/*---------------------------------------------------------------------------*/
/*  Class                                                                    */
/*---------------------------------------------------------------------------*/

/*---------------------------------------------------------------------------*/
/*  Namespace declared - end                                                 */
/*---------------------------------------------------------------------------*/

}

