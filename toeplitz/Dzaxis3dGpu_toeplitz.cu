#include "hip/hip_runtime.h"
/**************************************************************************
   (C) Copyright 2010 The Board of Trustees of the University of Illinois.
   All rights reserved.

   Developed by:

                    IMPACT & MRFIL Research Groups
               University of Illinois, Urbana Champaign

   Permission is hereby granted, free of charge, to any person obtaining a copy
   of this software and associated documentation files (the "Software"), to
   deal with the Software without restriction, including without limitation the
   rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
   sell copies of the Software, and to permit persons to whom the Software is
   furnished to do so, subject to the following conditions:

   Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimers.

   Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimers in the documentation
   and/or other materials provided with the distribution.

   Neither the names of the IMPACT Research Group, MRFIL Research Group, the
   University of Illinois, nor the names of its contributors may be used to
   endorse or promote products derived from this Software without specific
   prior written permission.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
   OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
   CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
   THE SOFTWARE.
 *****************************************************************************/

/*****************************************************************************

    File Name   [Dzaxis3dGpu.cu]

    Revision    [0.1; Initial build; Fan Lam, Mao-Jing Fu, ECE UIUC]
    Date        [10/25/2010]

*****************************************************************************/

/*---------------------------------------------------------------------------*/
/*  Included library headers                                                 */
/*---------------------------------------------------------------------------*/

// System libraries
#include <stdio.h>
#include <string.h>
#include <assert.h>

// Project header files
#include <hipfft/hipfft.h>
#include "Dzaxis3dGpu_toeplitz.cuh"

#define Dzaxis_BLOCK_X      128
#define Dzaxis_BLOCK_X_2D   16
#define Dzaxis_BLOCK_Z_2D   16

__global__ void Dzaxis3dGpuKernel1(
    hipfftComplex *s, hipfftComplex *p,
    const unsigned int num_row, const unsigned int num_col,
    const unsigned int num_dep);

__global__ void Dzaxis3dGpuKernel2(
    hipfftComplex *s, hipfftComplex *p,
    const unsigned int num_row, const unsigned int num_col,
    const unsigned int num_dep);

    void
Dzaxis3dGpu( matrix_t &s, matrix_t &p )
{
    const unsigned int num_row = p.dims[0];// y dimension
    const unsigned int num_col = p.dims[1];// x dimension
    const unsigned int num_dep = p.dims[2];// z dimension
    const unsigned int num = num_row*num_col*num_dep;

    int num_blocks_x = ceil((float) num_col / (float) Dzaxis_BLOCK_X);
    int num_blocks_x_2d = ceil((float) num_col / (float) Dzaxis_BLOCK_X_2D);
    int num_blocks_z_2d = ceil((float) num_dep / (float) Dzaxis_BLOCK_Z_2D);

    assert( num_blocks_x <= 65535 );
    assert( num_blocks_x_2d * num_blocks_z_2d <= 65535 );

    Dzaxis3dGpuKernel1 <<<dim3(num_blocks_x_2d, num_blocks_z_2d),
                          dim3(Dzaxis_BLOCK_X_2D, Dzaxis_BLOCK_Z_2D)>>>
        (s.device, p.device, num_row, num_col, num_dep);

    Dzaxis3dGpuKernel2 <<<dim3(num_blocks_x), dim3(Dzaxis_BLOCK_X)>>>
        (s.device, p.device, num_row, num_col, num_dep);
}

__global__ void Dzaxis3dGpuKernel1(
    hipfftComplex *s, hipfftComplex *p,
    const unsigned int num_row, const unsigned int num_col,
    const unsigned int num_dep)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;// x-axis
    int z = blockIdx.y * blockDim.y + threadIdx.y;// z-axis

    if( (z < (num_dep-1)) && (x < num_col) )
    {
        for(int y=0;y<num_row;y++) {
           // LHS(y,x,z) = RHS(y,x,z) - RHS(y,x,z+1)
           const unsigned int y_x_z = y*num_col*num_dep + x*num_dep + z;
           const unsigned int y_x_z1 = y*num_col*num_dep + x*num_dep + (z+1);
           s[y_x_z]REAL = p[y_x_z]REAL - p[y_x_z1]REAL;
           s[y_x_z]IMAG = p[y_x_z]IMAG - p[y_x_z1]IMAG;
        }
   }
}

__global__ void Dzaxis3dGpuKernel2(
    hipfftComplex *s, hipfftComplex *p,
    const unsigned int num_row, const unsigned int num_col,
    const unsigned int num_dep)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;// x axis

    if(x<num_col) 
    {
        for(int y=0;y<num_row;y++) {
           // LHS(y,x,num_dep-1) = RHS(y,x,num_dep-1) - RHS(y,x,0)
           const unsigned int y_x_num_dep_1 = y*num_col*num_dep + x*num_dep + (num_dep-1);
           const unsigned int y_x_zero = y*num_col*num_dep + x*num_dep;
           s[y_x_num_dep_1]REAL = p[y_x_num_dep_1]REAL - p[y_x_zero]REAL;
           s[y_x_num_dep_1]IMAG = p[y_x_num_dep_1]IMAG - p[y_x_zero]IMAG;
        }
    }
}
